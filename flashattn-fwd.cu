#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define data_t float


// forward kernel
__global__
void flash2_forward_kernel(
    const data_t* Q, const data_t* K, const data_t* V, const int N, const int d,
    const int Tc, const int Tr, const int Bc, const int Br, const data_t softmax_scale,
    data_t* L, data_t* O
){
    int tx  = threadIdx.x;
    int ntx = blockDim.x;

    int bx = blockIdx.x; // batch index
    int by = blockIdx.y; // head  index

    // offset into Q,K,V,O,l,m
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);
    int L_offset   = (bx * gridDim.y * N) + (by * N);

    // Define SRAM for Q, K, V, S
    extern __shared__ data_t sram[];
    int tile_size_Q  = Br * d;
    int tile_size_KV = Bc * d;
    data_t* Qi = sram;
    data_t* Kj = Qi + tile_size_Q;
    data_t* Vj = Kj + tile_size_KV;
    data_t* SP = Vj + tile_size_KV;

    data_t m[Br];
    data_t l[Br];

    for (int i = 0; i < Tr; i++){

        // load Qi
        for (int r = 0; r < Br; r += ntx){
            if (tx + r < Br){
                for (int x = 0; x < d; x++){
                    Qi[(tx + r) * d + x] = Q[qkv_offset + tile_size_Q * i + (tx + r) * d + x];
                }
            }
        }

        m[tx] = -INFINITY;
        l[tx] = 0;

        // __syncthreads();

        for (int j = 0; j < Tc; j++){

            // load Kj, Vj, m, l
            for (int c = 0; c < Bc; c += ntx){
                if (tx + c < Bc){
                    for (int x = 0; x < d; x++){
                        Kj[(tx + c) * d + x] = K[qkv_offset + tile_size_KV * j + (tx + c) * d + x];
                        Vj[(tx + c) * d + x] = V[qkv_offset + tile_size_KV * j + (tx + c) * d + x];
                    }
                }
            }

            __syncthreads();

            data_t row_m_prev = m[tx];
            data_t row_l_prev = l[tx];

            // compute SP = QK^T, row_m = rowmax(S)
            data_t row_m = -INFINITY;
            for (int rc = 0; rc < Bc * Br; rc += ntx){
                if (tx + rc < Bc * Br){
                    int y = rc / Bc;
                    int x = rc % Bc;
                    data_t sum = 0;
                    for (int z = 0; z < d; z++){
                        sum += Qi[y * d + z] * Kj[x * d + z];
                    }

                    sum *= softmax_scale;
                    SP[y * Bc + x] = sum;
                    if (sum > row_m){
                        row_m = sum;
                    }
                }
            }

            __syncthreads();

            // compute row_l, S -> P
            data_t row_l = 0;
            for (int r = 0; r < Br; r += ntx){
                if (tx + r < Br){
                    for (int x = 0; x < d; x++){
                        SP[(tx + r) * Bc + x] = __expf(SP[(tx + r) * Bc + x] - row_m);
                        row_l += SP[(tx + r) * Bc + x];}
                }
            }

            // compute new m, l
            data_t row_m_new = max(row_m, row_m_prev);

            data_t row_m_pre_new   = __expf(row_m_prev - row_m_new);
            data_t row_m_block_new = __expf(row_m - row_m_new);
            
            data_t row_l_new = row_m_pre_new * row_l_prev + row_m_block_new * row_l;

            // compute O, l, m
            for (int rd = 0; rd < Br * d; rd += ntx){
                if (tx + rd < Br * d){
                    int r = rd / d;
                    int x = rd % d;
                    data_t pv = 0; // Pij * Vj
                    for (int y = 0; y < Bc; y++){
                        pv += SP[r * Bc + y] * Vj[y * d + x];
                    }
                    O[qkv_offset + tile_size_Q * i + r * d + x] = O[qkv_offset + tile_size_Q * i + r * d + x] * row_m_pre_new + pv;
                }
            }

            for (int r = 0; r < Br; r += ntx){
                if (tx + r < Br){
                    m[tx + r] = row_m_new;
                    l[tx + r] = row_l_new;
                }
            }
        }

        // write O, L to HBM
        for (int rd = 0; rd < Br * d; rd += ntx){
            if (tx + rd < Br * d){
                int r = rd / d;
                int x = rd % d;
                O[qkv_offset + tile_size_Q * i + r * d + x] = O[qkv_offset + tile_size_Q * i + r * d + x] / l[tx];
            }
        }

        for (int r = 0; r < Br; r += ntx){
            if (tx + r < Br){
                L[L_offset + i * Br + tx] = l[tx];
            }
        }
        
        // __syncthreads();
    }
}



// forward function
torch :: Tensor forward(
    torch :: Tensor Q, torch :: Tensor K, torch :: Tensor V
) {
    const int Bc = 32;
    const int Br = 32;

    const int B  = Q.size(0);
    const int nh = Q.size(1);
    const int N  = Q.size(2);
    const int d  = Q.size(3);

    const int Tr = (N + Br - 1) / Br;
    const int Tc = (N + Bc - 1) / Bc;
    const float softmax_scale = 1.0f / sqrtf(d);

    auto O = torch::zeros_like(Q);
    auto L = torch::zeros({B, nh, N});

    torch::Device device(torch::kCUDA);
    L = L.to(device);

    const int sram_size = 2* Bc * d * sizeof(data_t) + Br * d * sizeof(data_t) + Br * Bc * sizeof(data_t);
    int max_sram_size;

    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory per block: %d, Requested shared memory: %d \n", max_sram_size, sram_size);

    dim3 grid(Tr, nh);
    dim3 block(Br);

    flash2_forward_kernel<<<grid, block, sram_size>>>(Q.data_ptr<data_t>(), K.data_ptr<data_t>(), V.data_ptr<data_t>(), N, d, Tc, Tr, Bc, Br, softmax_scale, L.data_ptr<data_t>(), O.data_ptr<data_t>());

    return O;

}




